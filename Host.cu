#include "hip/hip_runtime.h"
#include <cstdio>
#include <fstream>
#include <unordered_map>
#include <iostream>
#include <sstream>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <limits>
#include <future>
#include "./lda/LDA.cpp"
#include <iomanip>
#include <algorithm>
#include <string>
#include <vector>
#include <map>
#include <cmath>
#include "./cuda_util/CudaUtil.h"
#include "./hyperblock_generation/MergerHyperBlock.cuh"
#include "./hyperblock/HyperBlock.h"
#include "./interval_hyperblock/IntervalHyperBlock.h"
#include "./knn/Knn.h"
#include "./screen_output/PrintingUtil.h"
#include "./data_utilities/DataUtil.h"
#include "./simplifications/Simplifications.h"
using namespace std;

#ifdef _WIN32
    const string PATH_SEPARATOR = "\\";
#else
    const string PATH_SEPARATOR = "/";
#endif

#define LDA_ORDERING true

int NUM_CLASSES;   // Number of classes in the dataset
int NUM_POINTS;    // Total number of points in the dataset
int FIELD_LENGTH;  // Number of attributes in the dataset
int COMMAND_LINE_ARGS_CLASS = -1; // used for when we are splitting up generation one class per machine. This lets us run on many computers at once.

map<string, int> CLASS_MAP;
map<string, int> CLASS_MAP_TESTING;

map<int, string> CLASS_MAP_INT;
map<int, string> CLASS_MAP_TESTING_INT;

/**
* We generate a confusion matrix, but allow for points to fall into multiple blocks at a time
* that is why we go through blocks on outerloop and whole dataset on the inside.
*/
vector<vector<long>> testAccuracyOfHyperBlocks(vector<HyperBlock>& hyperBlocks, vector<vector<vector<float>>> &testSet){

  	// Keep track of which points were never inside of a block, when a point is classifed we increment the map internal vectors correct positon
    // there should be CLASS_NUM unordered_maps or just hashmaps, in each will hold a vector<point_index, vector<int> of len(class_num)>
    vector<unordered_map<int, vector<int>>> pointsNotClassified(CLASS_MAP.size());

    // Go through each class
    for(int cls = 0; cls < NUM_CLASSES; cls++){
        // Put the index of each point in each class into a set, this is how we will track which points were never classified.
        for(int j = 0; j < testSet[cls].size(); j++){
            pointsNotClassified[cls][j] = vector<int>(NUM_CLASSES);
        }
    }

	// Make a n x n matrix for the confusion matrix
	vector<vector<long>> ultraConfusionMatrix(NUM_CLASSES, vector<long>(NUM_CLASSES, 0));
    vector<vector<long>> regularConfusionMatrix(NUM_CLASSES, vector<long>(NUM_CLASSES, 0));

    bool anyPointWasInside = false;

    // Go through all the blocks
	for(int hb = 0; hb < hyperBlocks.size(); hb++){
        HyperBlock& currBlock = hyperBlocks[hb];
        // Go through all the classes in the testSet
		for(int cls = 0; cls < NUM_CLASSES; cls++){
            // go through all the points in a clases
        	for(int pnt = 0; pnt < testSet[cls].size(); pnt++){
           		const vector<float>& point = testSet[cls][pnt];

                if(currBlock.inside_HB(point.size(), point.data())){

					ultraConfusionMatrix[cls][currBlock.classNum]++;

                    // Go to the actual class, to the right points entry, and increment the "predicted" class (the hb it was in).
                    pointsNotClassified[cls][pnt][currBlock.classNum]++;
                }
        	}
     	}
    }

    // Lets count how many points fell into blocks of multiple classes
    for(int i = 0; i < NUM_CLASSES; i++){
       int numPointsInMultipleClasses = 0;
       int numPointsInNoBlocks = 0;

       // Go through all the points in a class.
       for(int pnt = 0; pnt < testSet[i].size(); pnt++){
           char in = 0;

           // Go through the classification vector for the point
           for(int cls = 0; cls < NUM_CLASSES; cls++){
               if(pointsNotClassified[i][pnt][cls] > 0){
                  in++;
               }

               // Means it fell into multiple of the same.
               if(in > 1){
                   break;
               }
           }

           if(in > 1) {
               numPointsInMultipleClasses++;
           }

           if(in == 0) numPointsInNoBlocks++;
       }

       cout << "CLASS: " << CLASS_MAP_INT[i] << "NUM POINTS IN MULTIPLE CLASSES BLOCKS: " << numPointsInMultipleClasses << endl;
       cout << "CLASS: " << CLASS_MAP_INT[i] << "NUM POINTS IN NO BLOCKS: " << numPointsInNoBlocks << endl;
    }

    vector<vector<vector<float>>> unclassifiedPointVec(NUM_CLASSES, vector<vector<float>>()); // [class][pointIdx][attr]

    // Lets count how many points fell into blocks of multiple classes
    for(int i = 0; i < NUM_CLASSES; i++){
        // Go through all the points in a class.
        for(int pnt = 0; pnt < testSet[i].size(); pnt++){
            int majorityClass = -1;
            int max = 0;

            // Go through the classification vector for the point
            for(int cls = 0; cls < NUM_CLASSES; cls++){
                if(pointsNotClassified[i][pnt][cls] > max){
                   max = pointsNotClassified[i][pnt][cls];
                   majorityClass = cls;
                }
            }

            // The majority was the one they are actually predicted to be in
            if(majorityClass != -1){
                regularConfusionMatrix[i][majorityClass]++;

            }else{
                // Put the point that wasn't classified into the vector to go to Knn
                unclassifiedPointVec[i].push_back(testSet[i][pnt]);
            }
        }
    }

    cout << "\n\n\n\n" << endl;
    cout << "============================ REGULAR CONFUSION MATRIX ==================" << endl;
    PrintingUtil::printConfusionMatrix(regularConfusionMatrix, NUM_CLASSES, CLASS_MAP_INT);
    cout << "============================ END CONFUSION MATRIX ======================" << endl;

	cout << "Any point was inside" << anyPointWasInside <<  endl;

    cout << "\n\n\n\n" << endl;
    cout << "============================ K-NN CONFUSION MATRIX ==================" << endl;
    int k = 1;
    vector<vector<long>> secondConfusionMatrix = Knn::kNN(unclassifiedPointVec, hyperBlocks, k, NUM_CLASSES);
     PrintingUtil::printConfusionMatrix(secondConfusionMatrix, NUM_CLASSES, CLASS_MAP_INT);
    cout << "============================ END K-NN MATRIX ======================" << endl;
    for (int i = 0; i < NUM_CLASSES; i++) {
        for (int j = 0; j < NUM_CLASSES; j++) {
            regularConfusionMatrix[i][j] = regularConfusionMatrix[i][j] + secondConfusionMatrix[i][j];
        }
    }

    cout << "\n\n\n\n" << endl;
    cout << "============================ DISTINCT POINT CONFUSION MATRIX ==================" << endl;
    PrintingUtil::printConfusionMatrix(regularConfusionMatrix, NUM_CLASSES, CLASS_MAP_INT);
    cout << "============================ END DISTINCT POINT MATRIX ======================" << endl;
    cout << "\n\n\n\n" << endl;

    return ultraConfusionMatrix;
}

// This function computes the LDA ordering for a given training dataset.
// It sets up the bestVectors, bestVectorsIndexes, and eachClassBestVectorIndex.
// best vectors is the weights of each coefficient from the LDF function
// bestVectorsIndexes is just the indexes that correspond to those weights from the function, since we are sorting them
// eachClassBestVectorIndex is the one best attribute for each class, we sort by this when generating blocks, and it helps a bit.
void computeLDAOrdering(const vector<vector<vector<float>>>& trainingData, vector<vector<float>>& bestVectors, vector<vector<int>>& bestVectorsIndexes, vector<int>& eachClassBestVectorIndex) {
    // Run LDA on the training data.
    bestVectors = linearDiscriminantAnalysis(trainingData);

    // Resize our index containers.
    bestVectorsIndexes.assign(NUM_CLASSES, vector<int>(FIELD_LENGTH, 0));
    eachClassBestVectorIndex.assign(NUM_CLASSES, 0);

    // For each class, initialize the indexes and then sort (if desired)
    // and determine the index with the largest absolute LDA coefficient.
    for (int i = 0; i < NUM_CLASSES; i++) {
        // Populate with initial indices: 0, 1, 2, ... FIELD_LENGTH - 1.
        for (int j = 0; j < FIELD_LENGTH; j++) {
            bestVectorsIndexes[i][j] = j;
        }

#ifdef LDA_ORDERING
        // Optionally sort the indexes for class i based on the absolute value of the LDA coefficients.
        sort(bestVectorsIndexes[i].begin(), bestVectorsIndexes[i].end(),
             [&](int a, int b) {
                 return fabs(bestVectors[i][a]) < fabs(bestVectors[i][b]);
             });
#endif
        // Find the index (from bestVectorsIndexes) corresponding to the largest absolute LDA coefficient.
        // We use the values in bestVectors[i] for comparison.
        auto it = max_element(bestVectorsIndexes[i].begin(), bestVectorsIndexes[i].end(),
                              [&](int a, int b) {
                                  return fabs(bestVectors[i][a]) < fabs(bestVectors[i][b]);
                              });
        eachClassBestVectorIndex[i] = distance(bestVectorsIndexes[i].begin(), it);
    }
}

void runKFold(vector<vector<vector<float>>> &dataset) {
    if (dataset.empty()) {
        cout << "Please enter a training dataset before using K Fold validation" << endl;
        return;
    }

    cout << "Please Enter a K value:\t";
    int k;
    cin >> k;

    // Clear the newline from the input buffer.
    cin.ignore(numeric_limits<streamsize>::max(), '\n');

    if (cin.fail() || k < 2) {
        cout << "Error: Invalid input. Please enter a valid integer greater than 1." << endl;
        // Clear the error state and ignore any remaining input.
        cin.clear();
        cin.ignore(numeric_limits<streamsize>::max(), '\n');
        return;
    }

    vector<vector<vector<vector<float>>>> kFolds = DataUtil::splitDataset(dataset, k);


    // generate blocks with a training set which is all folds except i. using i as the test dataset.
    for (int i = 0; i < k; i++) {

        // trainingData will store all folds except the i-th as training data.
        vector<vector<vector<float>>> trainingData(NUM_CLASSES);

        // Loop through all folds except i and accumulate points by class.
        for (int fold = 0; fold < k; fold++) {
            if (fold == i) continue; // skip test fold

            for (int cls = 0; cls < NUM_CLASSES; cls++) {
                // Append all points from kFolds[fold][cls] to trainingData[cls]
                trainingData[cls].insert(trainingData[cls].end(), kFolds[fold][cls].begin(), kFolds[fold][cls].end());
            }
        }

        // The test dataset for this iteration is simply fold i.
        vector<vector<vector<float>>> testData = kFolds[i];

        // now that our data is set up with training and testing, we simply do business as usual. we are going to do our LDA on the train data, then just do our block generation and simplification
        // Run LDA on the training data.
        vector<vector<float>>bestVectors;

        // Initialize indexes for each class
        vector<vector<int>> bestVectorsIndexes = vector<vector<int> >(NUM_CLASSES, vector<int>(FIELD_LENGTH, 0));
        vector<int> eachClassBestVectorIndex = vector<int>(NUM_CLASSES);

        computeLDAOrdering(trainingData, bestVectors, bestVectorsIndexes, eachClassBestVectorIndex);


        cout << "----------------------------FOLD " << (i + 1) << " RESULTS----------------------------------" << endl;

        // ------------------------------------------
        // GENERATING BLOCKS BUSINESS AS USUAL
        vector<HyperBlock> hyperBlocks;

        IntervalHyperBlock::generateHBs(trainingData, hyperBlocks, eachClassBestVectorIndex, FIELD_LENGTH, COMMAND_LINE_ARGS_CLASS);
        cout << "HYPERBLOCK GENERATION FINISHED!" << endl;
        cout << "WE FOUND " << hyperBlocks.size() << " HYPERBLOCKS!" << endl;

        vector<int> result = Simplifications::runSimplifications(hyperBlocks, trainingData, bestVectorsIndexes);
        int totalPoints = 0;
        for (const auto &c : trainingData)
            totalPoints += c.size();
        cout << "After removing useless blocks we have: " << result[1] << " clauses\n";
        cout << "Ran simplifications: " << result[0] << " Times" << endl;
        cout << "We had: " << totalPoints << " points" << endl;

        testAccuracyOfHyperBlocks(hyperBlocks, testData);
    } // end of one train/test loop
}

// -------------------------------------------------------------------------
// Asynchronous mode: run when argc >= 2
int runAsync(int argc, char* argv[]) {
    // Local variables for async mode
    string normalizedSaveFile;
    string hyperBlocksImportFileName;
    string trainingDataFileName;
    string testingDataFileName;
    string hyperBlocksExportFileName;

    // 3-D DATASETS
    vector<vector<vector<float>>> testData;
    vector<vector<vector<float>>> trainingData;

    // Normalization vectors (will be resized later)
    vector<float> minValues;
    vector<float> maxValues;

    // Store our HyperBlocks
    vector<HyperBlock> hyperBlocks;

    // Ultra confusion matrix
    vector<vector<long>> ultraConfusionMatrix;

    // Variables to be set by LDA

    if (argc > 3) {
        cout << "TOO MANY ARGUMENTS!" << endl;
        exit(1);
    }

    if (argc == 3) {
        // Set a global or externally-declared variable
        COMMAND_LINE_ARGS_CLASS = stoi(argv[2]);
        cout << "Running on class index " << COMMAND_LINE_ARGS_CLASS << endl;
    }

    // Process training data from file provided as first argument
    trainingData = DataUtil::dataSetup(argv[1], CLASS_MAP, CLASS_MAP_INT);
    cout << "NUM ATTRIBUTES : " << FIELD_LENGTH << endl;
    cout << "NUM CLASSES : " << NUM_CLASSES << endl;

    // Resize normalization vectors based on FIELD_LENGTH
    minValues.assign(FIELD_LENGTH, numeric_limits<float>::infinity());
    maxValues.assign(FIELD_LENGTH, -numeric_limits<float>::infinity());

    DataUtil::findMinMaxValuesInDataset(trainingData, minValues, maxValues, FIELD_LENGTH);
    DataUtil::minMaxNormalization(trainingData, minValues, maxValues, FIELD_LENGTH);

    // Run LDA on the training data.
    vector<vector<float>>bestVectors;
    // Initialize indexes for each class
    vector<vector<int>> bestVectorsIndexes = vector<vector<int> >(NUM_CLASSES, vector<int>(FIELD_LENGTH, 0));
    vector<int> eachClassBestVectorIndex = vector<int>(NUM_CLASSES);
    computeLDAOrdering(trainingData, bestVectors, bestVectorsIndexes, eachClassBestVectorIndex);

    IntervalHyperBlock::generateHBs(trainingData, hyperBlocks, eachClassBestVectorIndex, FIELD_LENGTH, COMMAND_LINE_ARGS_CLASS);
    cout << "HYPERBLOCK GENERATION FINISHED!" << endl;
    cout << "WE FOUND " << hyperBlocks.size() << " HYPERBLOCKS!" << endl;

    vector<int> result = Simplifications::runSimplifications(hyperBlocks, trainingData, bestVectorsIndexes);
    int totalPoints = 0;
    for (const auto &c : trainingData)
        totalPoints += c.size();
    cout << "After removing useless blocks we have: " << result[1] << " clauses\n";
    cout << "Ran simplifications: " << result[0] << " Times" << endl;
    cout << "We had: " << totalPoints << " points\n";

     DataUtil::saveBasicHBsToCSV(hyperBlocks, "AsyncBlockOutput", FIELD_LENGTH);
    return 0;
}

// -------------------------------------------------------------------------
// Interactive mode: run when argc < 2
void runInteractive() {
    // Local variables for interactive mode
    string normalizedSaveFile;
    string hyperBlocksImportFileName;
    string trainingDataFileName;
    string testingDataFileName;
    string hyperBlocksExportFileName;
    vector<vector<vector<float>>> testData;
    vector<vector<vector<float>>> trainingData;

    vector<float> minValues;
    vector<float> maxValues;

    vector<HyperBlock> hyperBlocks;

    vector<vector<long>> ultraConfusionMatrix;

    vector<vector<float>> bestVectors;
    vector<vector<int>> bestVectorsIndexes;
    vector<int> eachClassBestVectorIndex;

    bool running = true;
    int choice;
    while (running) {
        PrintingUtil::displayMainMenu();
        cin >> choice;
        cin.clear();
        cin.ignore(numeric_limits<streamsize>::max(), '\n');

        switch (choice) {
            case 1: { // IMPORT TRAINING DATA
                cout << "Enter training data filename: " << endl;
                system("ls datasets");
                getline(cin, trainingDataFileName);

                string fullPath = "datasets" + string(PATH_SEPARATOR) + trainingDataFileName;

                CLASS_MAP.clear();
                CLASS_MAP_INT.clear();

                trainingData = DataUtil::dataSetup(fullPath.c_str(), CLASS_MAP, CLASS_MAP_INT);

                // Resize normalization vectors based on FIELD_LENGTH
                minValues.assign(FIELD_LENGTH, numeric_limits<float>::infinity());
                maxValues.assign(FIELD_LENGTH, -numeric_limits<float>::infinity());
                DataUtil::findMinMaxValuesInDataset(trainingData, minValues, maxValues, FIELD_LENGTH);
                DataUtil::minMaxNormalization(trainingData, minValues, maxValues, FIELD_LENGTH);

                computeLDAOrdering(trainingData, bestVectors, bestVectorsIndexes, eachClassBestVectorIndex);


                PrintingUtil::waitForEnter();
                break;
            }
            case 2: { // IMPORT TESTING DATA
                cout << "Enter testing data filename: " << endl;
                system("ls datasets");
                getline(cin, testingDataFileName);
                string fullPath = "datasets" + string(PATH_SEPARATOR) + testingDataFileName;

                // clear these two maps to prevent issues when using a second test set.
                CLASS_MAP_TESTING.clear();
                CLASS_MAP_TESTING_INT.clear();

                testData = DataUtil::dataSetup(fullPath, CLASS_MAP_TESTING, CLASS_MAP_TESTING_INT);

                // Normalize and reorder testing data as needed.
                DataUtil::normalizeTestSet(testData, minValues, maxValues, FIELD_LENGTH);
                testData = DataUtil::reorderTestingDataset(testData, CLASS_MAP, CLASS_MAP_TESTING);
                PrintingUtil::waitForEnter();
                break;
            }
            case 3: { // SAVE NORMALIZED TRAINING DATA
                cout << "Enter the file to save the normalized training data to: " << endl;
                getline(cin, normalizedSaveFile);
                DataUtil::saveNormalizedVersionToCsv(normalizedSaveFile, trainingData);
                cout << "Saved normalized training data to: " << normalizedSaveFile << endl;
                PrintingUtil::waitForEnter();
                break;
            }
            case 4: { // IMPORT EXISTING HYPERBLOCKS
                cout << "Enter existing hyperblocks file name: " << endl;
                getline(cin, hyperBlocksImportFileName);
                hyperBlocks = DataUtil::loadBasicHBsFromCSV(hyperBlocksImportFileName);
                cout << "HyperBlocks imported from file " << hyperBlocksImportFileName << " successfully" << endl;
                PrintingUtil::waitForEnter();
                break;
            }
            case 5: { // EXPORT HYPERBLOCKS
                cout << "Enter the file to save HyperBlocks to: " << endl;
                getline(cin, hyperBlocksExportFileName);
                DataUtil::saveBasicHBsToCSV(hyperBlocks, hyperBlocksExportFileName, FIELD_LENGTH);
                break;
            }
            case 6: { // GENERATE NEW HYPERBLOCKS
                if (trainingData.empty()) {
                    cout << "\nError: Please import training data first." << endl;
                    PrintingUtil::waitForEnter();
                } else {
                    hyperBlocks.clear();
                    IntervalHyperBlock::generateHBs(trainingData, hyperBlocks, eachClassBestVectorIndex, FIELD_LENGTH, COMMAND_LINE_ARGS_CLASS);
                }
                cout << "Finished Generating HyperBlocks" << endl;
                PrintingUtil::waitForEnter();
                break;
            }
            case 7: {
                // SIMPLIFY HYPERBLOCKS
                vector<int> result = Simplifications::runSimplifications(hyperBlocks, trainingData, bestVectorsIndexes);
                int totalPoints = 0;

                for (const auto &c : trainingData) totalPoints += c.size();

                cout << "After removing useless blocks we have: " << result[1] << " clauses\n";
                cout << "We got a final total of: " << hyperBlocks.size() << " blocks." << endl;
                cout << "We had: " << totalPoints << " points of training data\n";
                PrintingUtil::waitForEnter();
                break;
            }
            case 8: { // TEST HYPERBLOCKS ON DATASET
                cout << "Testing hyperblocks on testing dataset" << endl;
                ultraConfusionMatrix = testAccuracyOfHyperBlocks(hyperBlocks, testData);
                PrintingUtil::waitForEnter();
                break;
            }
            case 9: {
                runKFold(trainingData);
                PrintingUtil::waitForEnter();
                break;
            }
            case 10: {
                running = false;
                break;
            }
            default: {
                cout << "\nInvalid choice. Please try again." << endl;
                PrintingUtil::waitForEnter();
                break;
            }
        }
    }
}

// -------------------------------------------------------------------------
// Main entry point: choose mode based on argc.
int main(int argc, char* argv[]) {

    // Command line input mode, allows you to specify in command line what to do
    if (argc >= 2)
        return runAsync(argc, argv);

    // Interactive input loop, options to import data, train, test, save, etc
    runInteractive();
    return 0;
}