#include "hip/hip_runtime.h"
#include <cstdio>
#include <vector>
#include <string>
#include <map>
#include <fstream>
#include <iostream>
#include <sstream>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <limits>
#include <future>
#include "CudaUtil.h"
#include "HyperBlock.h"
#include "HyperBlockCuda.cuh"
#include <chrono>
#include <omp.h>
#include <iomanip>  // For setw
#include <algorithm> // For max
using namespace std;

int NUM_CLASSES;   // Number of classes in the dataset
int NUM_POINTS;    // Total number of points in the dataset
int FIELD_LENGTH;  // Number of attributes in the dataset
int COMMAND_LINE_ARGS_CLASS = -1;


map<string, int> CLASS_MAP_TRAINING;
map<string, int> CLASS_MAP_TESTING;


// Struct version of DataATTR xrecord
struct DataATTR {
    float value; // Value of one attribute of a point
    int classNum; // The class number of the point
    int classIndex; // The index of point within the class

    DataATTR(float val, int cls, int index) : value(val), classNum(cls), classIndex(index) {}
};

// Interval struct to make interval thing more understandable
struct Interval{
    int size;
    int start;
    int end;
    int attribute;

    Interval(int s, int st, int e, int a) : size(s), start(st), end(e), attribute(a) {}
};

Interval longestInterval(vector<DataATTR>& dataByAttribute, float accThreshold, vector<HyperBlock>& existingHB, int attr);
void removeValueFromInterval(vector<DataATTR>& dataByAttribute, Interval& intr, float value);
int skipValueInInterval(vector<DataATTR>& dataByAttribute, int i, float value);
bool checkIntervalOverlap(vector<DataATTR>& dataByAttribute, Interval& intr, int attr, vector<HyperBlock>& existingHB);
void merger_cuda(const vector<vector<vector<float>>>& dataWithSkips, const vector<vector<vector<float>>>& allData, vector<HyperBlock>& hyperBlocks);
void saveBasicHBsToCSV(const vector<HyperBlock>& hyperBlocks);
void print3DVector(const vector<vector<vector<float>>>& vec);


/***
* We want to go through the hyperBlocks that were generated and write them to a file.
*
*
* This print isn't caring about disjunctive blocks.
*/
void saveBasicHBsToCSV(const vector<HyperBlock>& hyperBlocks, const string& fileName){
	// Open file for writing
    ofstream file(fileName);
    if (!file.is_open()) {
        cerr << "Error opening file: " << fileName << endl;
        return;
    }

	// min1, min2, min3, ..., minN, max1, max2, max3, ..., maxN, class
	for (const auto& hyperBlock : hyperBlocks) {
        // Write minimums
        for (const vector<float>& min : hyperBlock.minimums) {
            file << min[0] << ",";
        }

        // Write maximums
        for (const vector<float>& max : hyperBlock.maximums) {
            file << max[0] << ",";
        }

        // Write the class number
        file << hyperBlock.classNum << "\n";
    }

    file.close();
}

///////////////////////// FUNCTIONS FOR intervalHyper IMPLEMENTATION /////////////////////////

 /**
     * Finds largest interval across all dimensions of a set of data.
     * @param dataByAttribute all data split by attribute
     * @param accThreshold accuracy threshold for interval
     * @param existingHB existing hyperblocks to check for overlap
     * @return largest interval
     */
vector<DataATTR> intervalHyper(vector<vector<DataATTR>>& dataByAttribute, float accThreshold, vector<HyperBlock>& existingHB){
    //cout << "Starting interval hyperblock" << endl;
    vector<future<Interval>> intervals;
    int attr = -1;
    Interval best(-1, -1, -1, -1);

   // Search each attribute
    for (int i = 0; i < dataByAttribute.size(); i++) {
        // Launch async task
        intervals.emplace_back(async(launch::async, longestInterval, ref(dataByAttribute[i]), accThreshold, ref(existingHB), i));
    }

    // Wait for results then find largest interval
    for(auto& future1 : intervals){
        Interval intr = future1.get();
        if(intr.size > 1 && intr.size > best.size){
            best.size = intr.size;
            best.start = intr.start;
            best.end = intr.end;
            best.attribute = intr.attribute;

            attr = intr.attribute;
        }
    }
	//cout << "Best.start: " << best.start << "  Best.end: " << best.end <<"  Best.size: " << best.size <<  "  Best.attribute: " << best.attribute << endl;

    // Construct ArrayList of data
    vector<DataATTR> longest;
    if(best.size != -1){
        for(int i = best.start; i <= best.end; i++){
          	//cout << "Data by attribute printing time: " << dataByAttribute[attr][i].classNum << " " << dataByAttribute[attr][i].classIndex << "\n";
            //cout << attr << endl;

            longest.push_back(dataByAttribute[attr][i]);
        }
    }
    //cout << "Finished interval hyperblock" << endl;

    return longest;
}

/**
 * Seperates data into seperate vecs by attribute
 */
vector<vector<DataATTR>> separateByAttribute(vector<vector<vector<float>>>& data){
    vector<vector<DataATTR>> attributes;

    // Go through the attribute columns
    for(int k = 0; k < FIELD_LENGTH; k++){
        vector<DataATTR> tmpField;

        // Go through the classes
        for(int i = 0; i < data.size(); i++){
            // Go through the points
            for(int j = 0; j < data[i].size(); j++){
                tmpField.push_back(DataATTR(data[i][j][k], i, j));
            }
        }
    
        // Sort data by value then add
        sort(tmpField.begin(), tmpField.end(), [](const DataATTR& a, const DataATTR& b) {
            return a.value < b.value;
        });
        attributes.push_back(tmpField);
    }

    return attributes;
}

/***
 * This will sort the array based on the "best" columns values
 * 
 * The columns themselves aren't moving, we are moving the points
 * based on the one columns values;
 */
void sortByColumn(vector<vector<float>>& classData, int colIndex) {
    sort(classData.begin(), classData.end(), [colIndex](const vector<float>& a, const vector<float>& b) {
        return a[colIndex] < b[colIndex];
    });
}

/***
 * Finds the longest interval in a sorted list of data by attribute.
 * @param dataByAttribute sorted data by attribute
 * @param accThreshold accuracy threshold for interval
 * @param existingHB existing hyperblocks to check for overlap
 * @param attr attribute to find interval on
 * @return longest interval
*/
Interval longestInterval(vector<DataATTR>& dataByAttribute, float accThreshold, vector<HyperBlock>& existingHB, int attr){
    //cout << "Started longest interval \n" << endl;

    Interval intr(1, 0, 0, attr);
    Interval max_intr(-1, -1, -1, attr);

    int n = dataByAttribute.size();
    float misclassified = 0;

    for(int i = 1; i < n; i++){
        // If current class matches with next
        if(dataByAttribute[intr.start].classNum == dataByAttribute[i].classNum){
            intr.size++;
        }
        else if( (misclassified+1) / intr.size > accThreshold){
            // ^ i think this is a poor way to check. but not changing rn for the translation from java
            misclassified++;
            intr.size++;
        }
        else{
            // Remove value from interval if accuracy is below threshold.
            if(dataByAttribute[i-1].value == dataByAttribute[i].value){
                // remove then skip overlapped values
                removeValueFromInterval(dataByAttribute, intr, dataByAttribute[i].value);
                i = skipValueInInterval(dataByAttribute, i, dataByAttribute[i].value);
            }

            // Update longest interval if it doesn't overlap
            if(intr.size > max_intr.size && checkIntervalOverlap(dataByAttribute, intr, attr, existingHB)){
                max_intr.start = intr.start;
                max_intr.end = intr.end;
                max_intr.size = intr.size;
                max_intr.attribute = attr;
            }

            // Reset curr interval
            intr.size = 1;
            intr.start = i;
            misclassified = 0;
        }
        intr.end = i;
    }

    // final check update longest interval if it doesn't overlap
    if(intr.size > max_intr.size && checkIntervalOverlap(dataByAttribute, intr, attr, existingHB)){
        max_intr.start = intr.start;
        max_intr.end = intr.end;
        max_intr.size = intr.size;
    }

    //cout << "Finished longest interval \n" << endl;

    return max_intr;
}


bool checkIntervalOverlap(vector<DataATTR>& dataByAttribute, Interval& intr, int attr, vector<HyperBlock>& existingHB){
    //cout << "Started check interval overlap\n" << endl;
    // interval range of vals
    float intv_min = dataByAttribute[intr.start].value;
    float intv_max = dataByAttribute[intr.end].value;
   
    /*
    *   check if interval range overlaps with any existing hyperblocks
    * to not overlap the interval maximum must be below all existing hyperblock minimums
    * or the interval minimum must be above all existing hyperblock maximums
    */
    for(const HyperBlock& hb : existingHB){
        if (!(intv_max < hb.minimums.at(attr).at(0) || intv_min > hb.maximums.at(attr).at(0))){
            return false;
        }
    }

    //cout << "Finished check interval overlap\n" << endl;

    // If unique return true
    return true;
}

//skipValueInInterval
int skipValueInInterval(vector<DataATTR>& dataByAttribute, int i, float value){
    //cout << "Starting skip value in interval\n" << endl;

    while(dataByAttribute[i].value == value){
        if(i < dataByAttribute.size() - 1){
            i++;
        }
        else{
            break;
        }
    }

    //cout << "Finished skip value in interval\n" << endl;

    return i;
}


//removeValueFromInterval
void removeValueFromInterval(vector<DataATTR>& dataByAttribute, Interval& intr, float value){
    //cout << "Starting remove value from intervals\n" << endl;
    while(dataByAttribute[intr.end].value == value){
        if(intr.end > intr.start){
            intr.size--;
            intr.end--;
        }
        else{
            intr.size = -1;
            break;
        }
    }
    //cout << "Finished remove value from intervals\n" << endl;
}

///////////////////////// END FUNCTIONS FOR intervalHyper IMPLEMENTATION /////////////////////////

void generateHBs(vector<vector<vector<float>>>& data, vector<HyperBlock>& hyperBlocks){
  	// "Started generating HBS\n" << endl;
    // Hyperblocks generated with this algorithm
    vector<HyperBlock> gen_hb;

    // Get data to create hyperblocks
    vector<vector<DataATTR>> dataByAttribute = separateByAttribute(data);
    vector<vector<DataATTR>> all_intv;

    // Create dataset without data from interval HyperBlocks
    vector<vector<vector<float>>> datum;
    vector<vector<vector<float>>> seed_data;
    vector<vector<int>> skips;
	// "Initialized datum, seed_data, skips\n" << endl;

    // Initially generate blocks

        //cout << "Starting while loop to generate hyperblocks\n";
		//cout << "dataByAttribute[0].size() = " << dataByAttribute[0].size() << endl;

        while(dataByAttribute[0].size() > 0){
			//cout << "Attempting to go into intervalHyper " << endl;

            vector<DataATTR> intv = intervalHyper(dataByAttribute, 100, gen_hb);
            all_intv.push_back(intv);
			//cout << "Pushed to back of all intervals" << endl;

            // if hyperblock is unique then add
            if(intv.size() > 1){
                //cout << "making hb and intv_data" << endl;
                vector<vector<vector<float>>> hb_data;
                vector<vector<float>> intv_data;


                // Add the points from real data that are in the intervals
                for(DataATTR& dataAttr : intv){
                    /*cout << "Trying to add a dataATTR. " << endl;

					if(dataAttr.classNum > 1 || dataAttr.classNum < 0){
                    	cout << "Invalid classNum: " << dataAttr.classNum << endl;
                        cout << "Value: " << dataAttr.value << endl;
                        continue;
                    }


                    if(dataAttr.classIndex > data[dataAttr.classNum].size() - 1 || dataAttr.classIndex < 0){
                    	cout << "Invalid class index: " << dataAttr.classIndex << endl;
                        continue;
                    }
					*/
                    intv_data.push_back(data[dataAttr.classNum][dataAttr.classIndex]);
                }

                //cout << "Made it past the points from real data thingy" << endl << endl;
                // add data and hyperblock
                hb_data.push_back(intv_data);
                //cout << "Added intv data to hb_data" << endl << endl;

                HyperBlock hb(hb_data, intv[0].classNum);
                //cout << "Made the hyperblock for this interval thing" << endl << endl;

                gen_hb.push_back(hb);
                //cout << "Added results from last intervalHyper" << endl << endl;
            }else{
                //cout << "Breaking because the intv size is < 1" << endl;
                break;
            }
        }

        // Add all hbs from gen_hb to hyperBlocks
        hyperBlocks.insert(hyperBlocks.end(), gen_hb.begin(), gen_hb.end());

        // All data: go through each class and add points from data
        for(const vector<vector<float>>& classData : data){
            datum.push_back(classData);
            seed_data.push_back(vector<vector<float>>());
            skips.push_back(vector<int>());
        }

        // find which data to skip
        for(const vector<DataATTR>& dataAttrs : all_intv){
            for(const DataATTR& dataAttr : dataAttrs){
                skips[dataAttr.classNum].push_back(dataAttr.classIndex);
            }
        }
        // Sort the skips
        for(vector<int>& skip : skips){
            sort(skip.begin(), skip.end());
        }

        for(int i = 0; i < data.size(); i++){
            for(int j = 0; j < data[i].size(); j++){
                if(skips[i].size() > 0){
                    if(j != skips[i][0]){
                        seed_data[i].push_back(data[i][j]);
                    }
                    else{
                      	// remove first element from skips[i]
                        skips[i].erase(skips[i].begin());
                    }
                }
                else{
                    seed_data[i].push_back(data[i][j]);
                }
            }
        }

        // Sort data by most important attribute
        for(int i = 0; i < datum.size(); i++){
            sortByColumn(datum[i], 278);
            sortByColumn(seed_data[i], 278);
        }

    // Call CUDA function.
    //cout << "Calling merger_cuda\n" << endl;

    try{
        //cout << "Printing interval hyperblocks:\n\n" << endl;
        //for(const auto& hb : hyperBlocks){
        //    for(const auto& min : hb.minimums){
        //        cout << min[0] << " ";
        //    }
        //   cout << endl;
		//
        //    for(const auto& max : hb.maximums){
        //        cout << max[0] << " ";
        //    }
        //    cout << endl;
        //}
        //cout << "End interval hyperblocks:\n\n" << endl;
		//cout << "DATUM BEING PASSED INTO MERGING:" << endl;
		//print3DVector(datum);

        //cout << "SEED DATA BEING PASSED INTO MERGING:" << endl;
		//print3DVector(seed_data);

        merger_cuda(seed_data, datum, hyperBlocks);
    }catch (exception e){
        cout << "Error in generateHBs: merger_cuda" << endl;
    }
}




void print3DVector(const vector<vector<vector<float>>>& vec) {
    for (int i = 0; i < vec.size(); i++) {
        cout << "Class " << i << ":" << endl;
        for (const auto& row : vec[i]) {
            cout << "  [";
            for (int j = 0; j < row.size(); j++) {
                cout << row[j];
                if (j < row.size() - 1) cout << ", ";
            }
            cout << "]" << endl;
        }
        cout << endl;  // Add spacing between classes
    }
}

/*  Returns a class seperated version of the dataset
 *  Each class has an entry in the outer vector with a 2-d vector of its points
 */
vector<vector<vector<float>>> dataSetup(const string filepath, map<string, int>& classMap) {
    // 3D vector: data[class][point][attribute]
    vector<vector<vector<float>>> data;

    ifstream file(filepath);
    if (!file.is_open()) {
        cerr << "Failed to open file " << filepath << endl;
        return data;
    }

    int classNum = 0;
    string line;
    // Ignore the header, can use later if needed
    getline(file, line);

    // Read through all rows of CSV
    while (getline(file, line)) {
        stringstream ss(line);
        string cell;
        vector<string> row;

        // Read the entire row, splitting by commas
        while (getline(ss, cell, ',')) {
            row.push_back(cell);
        }

        // Skip empty lines
        if (row.empty()) continue;

        string classLabel = row.back();
        row.pop_back();

        // Check if class exists, else create new entry
        if (classMap.count(classLabel) == 0) {
            classMap[classLabel] = classNum;
            data.push_back(vector<vector<float>>());
            classNum++;
        }

        int classIndex = classMap[classLabel];

        vector<float> point;
        for (const string& val : row) {
            try {
                point.push_back(stof(val));  // Convert to float and add to the point
            } catch (const invalid_argument&) {
                cerr << "Invalid value '" << val << "' in CSV" << endl;
                point.push_back(0.0f);  // Default to 0 if conversion fails
            }
        }

        // Add the points
        data[classIndex].push_back(point);
    }

    file.close();

    // Set global variables
    FIELD_LENGTH = data.empty() ? 0 : static_cast<int>(data[0][0].size());
    NUM_CLASSES = classNum;

    return data;
}


/* This needs to be a function to serialize hyperblocks.
 * take in 3-D vector that is the hyperblocks for each class
 * each class gets a dimension, with a 2-d vector for the HBs
 * assumes each row in the 2-D vector is 1 hyperblock
 * the first 1/2 of the row is the max's, the end is the mins.
 */
void saveHyperBlocksToFile(const string& filepath, const vector<vector<vector<float>>>& hyperBlocks) {
    ofstream file(filepath);

    if (!file.is_open()) {
        cerr << "Failed to open file: " << filepath << endl;
        return;
    }

    // Loop through each class (outermost vector)
    for (int classNum = 0; classNum < hyperBlocks.size(); classNum++) {
        // Loop through each hyperblock (2D vector)
        for (const auto& hyperblock : hyperBlocks[classNum]) {
            // Write hyperblock values
            for (int i = 0; i < hyperblock.size(); i++) {
                file << hyperblock[i];
                if (i < hyperblock.size()) file << ", ";
            }
            // Append the class index
            file << ", " << classNum << "\n";
        }
    }

    file.close();
    cout << "Hyperblocks saved to " << filepath << endl;
}


vector<HyperBlock> loadBasicHBsFromCSV(const string& fileName) {
    ifstream file(fileName);
    vector<HyperBlock> hyperBlocks;

    if (!file.is_open()) {
        cerr << "Error opening file: " << fileName << endl;
        return hyperBlocks;
    }

    string line;
    while (getline(file, line)) {
        stringstream ss(line);
        vector<vector<float>> minimums, maximums;
        string value;
        vector<float> temp_vals;

        while (getline(ss, value, ',')) {
            value.erase(0, value.find_first_not_of(" \t"));
            value.erase(value.find_last_not_of(" \t") + 1);

            if (!value.empty()) {
                temp_vals.push_back(stof(value));
            }
        }

        if (temp_vals.empty()) continue;

        int num_attributes = temp_vals.size() / 2;
        int classNum = static_cast<int>(temp_vals.back());
        temp_vals.pop_back(); // Remove classNum from the list

        for (int i = 0; i < num_attributes; ++i) {
            minimums.push_back({ temp_vals[i] });
            maximums.push_back({ temp_vals[i + num_attributes] });
        }

        hyperBlocks.emplace_back(maximums, minimums, classNum);
    }

    file.close();
    return hyperBlocks;
}




/**
* Find the min/max values in each column of data across the dataset.
* Can use this in normalization and also for making sure test set is normalized with
* the same values as the training set.
*/
void findMinMaxValuesInDataset(const vector<vector<vector<float>>>& dataset, vector<float>& minValues, vector<float>& maxValues) {
    // Step 1: Find min and max for each attribute
    for (const auto& class_data : dataset) {
        for (const auto& point : class_data) {
            for (int k = 0; k < FIELD_LENGTH; k++) {
                minValues[k] = min(minValues[k], point[k]);
                maxValues[k] = max(maxValues[k], point[k]);
            }
        }
    }
}




/**
* A function to normalize the test set using the given mins/maxes that were used to normalize the initial set
*/
void normalizeTestSet(vector<vector<vector<float>>>& testSet, const vector<float>& minValues, const vector<float>& maxValues) {
    if (testSet.empty()){
      cout << "Test set was empty when trying to normalize" << endl;
      return;
	}

    // Print out the min and max values first 20
    for (int i = 0; i < 30; i++) {
      cout << minValues[i] << endl;
    }
    cout << endl;

    cout << "Maxes" << endl;


    for (int i = 0; i < 30; i++) {
      cout << maxValues[i] << ",";
    }
    cout << endl;

    for (auto& class_data : testSet) {
        for (auto& point : class_data) {
            for (int k = 0; k < FIELD_LENGTH; k++) {
                if (maxValues[k] != minValues[k]) {
                    point[k] = (point[k] - minValues[k]) / (maxValues[k] - minValues[k]);
                } else {
                    point[k] = 0.5f;
                }
            }
        }
    }
}

void minMaxNormalization(vector<vector<vector<float>>>& dataset, const vector<float>& minValues, const vector<float>& maxValues) {
    //cout << "Starting min-max normalization\n" << endl;

    if (dataset.empty()) return;

    int num_classes = dataset.size();

    // Min and max values for each attribute
    //vector<float> minValues(FIELD_LENGTH, std::numeric_limits<float>::infinity());
    //vector<float> maxValues(FIELD_LENGTH, -std::numeric_limits<float>::infinity());

    // Step 1: Find min and max for each attribute
	//findMinMaxValuesInDataset(dataset, minValues, maxValues);

    // Step 2: Apply Min-Max normalization
    for (auto& class_data : dataset) {
        for (auto& point : class_data) {
            for (int k = 0; k < FIELD_LENGTH; k++) {
                // Avoid div/0
                if (maxValues[k] != minValues[k]) {
                    point[k] = (point[k] - minValues[k]) / (maxValues[k] - minValues[k]);
                } else {
                    //cout << "Column found with useless values" << endl;
                    point[k] = 0.5f;
                }
            }
        }
    }
    //cout << "Finished min-max normalization\n" << endl;
}

vector<bool> markUniformColumns(const vector<vector<vector<float>>>& data) {
     // cout << "Starting mark uniform columns\n" << endl;

    if (data.empty() || data[0].empty()) return vector<bool>(); // Handle edge case

    int numCols = data[0][0].size();
    vector<bool> removed(numCols, false);

    // Iterate through each column
    for (int col = 0; col < numCols; col++) {
        float referenceValue = data[0][0][col]; // Use first row of first class as reference
        bool allSame = true;

        // Check across all classes and all rows
        for (const auto& obj : data) {
            for (const auto& row : obj) {
                if (row[col] != referenceValue) {
                    allSame = false;
                    break;
                }
            }
            if (!allSame) break;
        }

        // If the column is uniform across all classes, mark it for removal
        if (allSame) {
            removed[col] = true;
        }
    }

   // cout << "Finished mark uniform columns\n" << endl;

    return removed;
}

// Source
void merger_cuda(const vector<vector<vector<float>>>& dataWithSkips, const vector<vector<vector<float>>>& allData, vector<HyperBlock>& hyperBlocks) {

    // Calculate total points
    int numPoints = 0;
    for (const auto& classData : allData) {
        numPoints += classData.size();
    }

    // Count blocks per class
    vector<int> numBlocksOfEachClass(NUM_CLASSES, 0);
    for (const auto& hb : hyperBlocks) {
        numBlocksOfEachClass[hb.classNum]++;
    }

    vector<vector<HyperBlock>> resultingBlocks(NUM_CLASSES);
    
    // get our device count, and max it so that the most we will use is numClasses if we have only 2 classes for example.
    /* MULTI GPU BUSINESS
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    deviceCount = std::min(deviceCount, NUM_CLASSES);


    // Process each class
    // we have our multithreading happen here at this level. set the device to class % deviceCount
    #pragma omp parallel for num_threads(deviceCount)
    */

    int PADDED_LENGTH = ((FIELD_LENGTH + 3) / 4) * 4;
    // Find best occupancy
    int sharedMemSize = 2 * PADDED_LENGTH * sizeof(float);
    int minGridSize, blockSize;
    hipError_t err = hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, mergerHyperBlocks, sharedMemSize, 0);
    if (err != hipSuccess) {
        printf("CUDA error in hipOccupancyMaxPotentialBlockSize: %s\n", hipGetErrorString(err));
        exit(-1);
    }


    int temp = 0;
    int goToClass = NUM_CLASSES;
    if (COMMAND_LINE_ARGS_CLASS != -1){
         temp = COMMAND_LINE_ARGS_CLASS;
         goToClass = COMMAND_LINE_ARGS_CLASS + 1;
    }

    for (int classN = temp; classN < goToClass; classN++) {
        
        // set our device based on class. this way even single threaded we use multiple GPUs
        // MORE MULTI GPU BUSINESS
        //hipSetDevice(classN % deviceCount);

        int totalDataSetSizeFlat = numPoints * PADDED_LENGTH;
        int sizeWithoutHBpoints = ((dataWithSkips[classN].size() + numBlocksOfEachClass[classN]) * PADDED_LENGTH);
        if (dataWithSkips[classN].empty()) {
            sizeWithoutHBpoints = numBlocksOfEachClass[classN] * PADDED_LENGTH;
        }

        // Compute grid size to cover all elements. we already know our ideal block size from before.
        int gridSize = ((sizeWithoutHBpoints / PADDED_LENGTH) + blockSize - 1) / blockSize;

        cout << "Grid size: " << gridSize << endl;
        cout << "Block size: " << blockSize << endl;
        cout << "Shared memory size: " << sharedMemSize << endl;

        // Allocate host memory
        vector<float> hyperBlockMinsC(sizeWithoutHBpoints);
        vector<float> hyperBlockMaxesC(sizeWithoutHBpoints);
        vector<float> combinedMinsC(sizeWithoutHBpoints);
        vector<float> combinedMaxesC(sizeWithoutHBpoints);
        vector<int> deleteFlagsC(sizeWithoutHBpoints / PADDED_LENGTH);

        int nSize = allData[classN].size();
        vector<float> pointsC(totalDataSetSizeFlat - (nSize * PADDED_LENGTH));

        // Fill hyperblock arrays
        int currentClassIndex = 0;
        for (int currentClass = 0; currentClass < dataWithSkips.size(); currentClass++) {
            for (const auto& point : dataWithSkips[currentClass]) {
                if (currentClass == classN) {
                    for (int attr = 0; attr < FIELD_LENGTH; attr++) {
                        //if (removed[attr]) continue;
                        hyperBlockMinsC[currentClassIndex] = point[attr];
                        hyperBlockMaxesC[currentClassIndex] = point[attr];
                        currentClassIndex++;
                    }
                    for (int leftOverAtt = FIELD_LENGTH; leftOverAtt < PADDED_LENGTH; leftOverAtt++) {
                        hyperBlockMinsC[currentClassIndex] = -std::numeric_limits<float>::infinity();
                        hyperBlockMaxesC[currentClassIndex] = std::numeric_limits<float>::infinity();
                        currentClassIndex++;
                    }
                }
            }
        }

        // Process other class points
        int otherClassIndex = 0;
        for (int currentClass = 0; currentClass < allData.size(); currentClass++) {
            if (currentClass == classN) continue;

            for (const auto& point : allData[currentClass]) {
                for (int attr = 0; attr < FIELD_LENGTH; attr++) {
                    pointsC[otherClassIndex++] = point[attr];
                }
                for (int leftOverAtt = FIELD_LENGTH; leftOverAtt < PADDED_LENGTH; leftOverAtt++) {
                    pointsC[otherClassIndex++] = -std::numeric_limits<float>::infinity();
                }
            }
        }

        // Add the existing blocks from intervalHyper
        for (auto it = hyperBlocks.begin(); it != hyperBlocks.end(); ++it) {
            if (it->classNum == classN) {
                for (int i = 0; i < it->minimums.size(); i++) {
                    //if (removed[i]) continue;
                    hyperBlockMinsC[currentClassIndex] = it->minimums[i][0];
                    hyperBlockMaxesC[currentClassIndex] = it->maximums[i][0];
                    currentClassIndex++;
                }
                for (int leftOverAtt = FIELD_LENGTH; leftOverAtt < PADDED_LENGTH; leftOverAtt++) {
                    hyperBlockMinsC[currentClassIndex] = -std::numeric_limits<float>::infinity();
                    hyperBlockMaxesC[currentClassIndex] = std::numeric_limits<float>::infinity();
                    currentClassIndex++;
                }
            }
        }

        // Allocate device memory
        float *d_hyperBlockMins, *d_hyperBlockMaxes, *d_combinedMins, *d_combinedMaxes, *d_points;
        int *d_deleteFlags, *d_mergable, *d_seedQueue, *d_writeSeedQueue;

        hipMalloc(&d_hyperBlockMins, sizeWithoutHBpoints * sizeof(float));
        hipMalloc(&d_hyperBlockMaxes, sizeWithoutHBpoints * sizeof(float));
        hipMalloc(&d_combinedMins, sizeWithoutHBpoints * sizeof(float));
        hipMalloc(&d_combinedMaxes, sizeWithoutHBpoints * sizeof(float));
        hipMalloc(&d_deleteFlags, (sizeWithoutHBpoints / PADDED_LENGTH) * sizeof(int));
        hipMemset(d_deleteFlags, 0, (sizeWithoutHBpoints / PADDED_LENGTH) * sizeof(int));

        hipMalloc(&d_points, pointsC.size() * sizeof(float));

        int numBlocks = hyperBlockMinsC.size() / PADDED_LENGTH;
        vector<int> seedQueue(numBlocks);
        for(int i = 0; i < numBlocks; i++){
            seedQueue[i] = i;
        }

        hipMalloc(&d_mergable, numBlocks * sizeof(int));
        hipMemset(d_mergable, 0, numBlocks * sizeof(int));
        hipMalloc(&d_seedQueue, numBlocks * sizeof(int));
        hipMalloc(&d_writeSeedQueue, numBlocks * sizeof(int));

        // Copy data to device
        hipMemcpy(d_hyperBlockMins, hyperBlockMinsC.data(), sizeWithoutHBpoints * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_hyperBlockMaxes, hyperBlockMaxesC.data(), sizeWithoutHBpoints * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_points, pointsC.data(), pointsC.size() * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_seedQueue, seedQueue.data(), numBlocks * sizeof(int), hipMemcpyHostToDevice);

        cout << "Launched a kernel for class: " << classN << endl;

        // funky wap to swap the readQueue and writeQueue
        int* queues[2] = {d_seedQueue, d_writeSeedQueue};
        for(int i = 0; i < numBlocks; i++){
            // swap between the two queues
            int* readQueue = queues[i & 1];
            int* writeQueue = queues[(i + 1) & 1];
            mergerHyperBlocksWrapper(
                i, 			// seednum
                readQueue,  // seedQueue
                numBlocks,  // number seed blocks
                PADDED_LENGTH,	// num attributes
                pointsC.size() / PADDED_LENGTH,	// num op class points
                d_points,						// op class points
                d_hyperBlockMins,				// mins
                d_hyperBlockMaxes,				// maxes
                d_deleteFlags,
                d_mergable,						// mergable flags
                gridSize,
                blockSize,
                sharedMemSize,
                d_combinedMins,
                d_combinedMaxes
            );
            hipDeviceSynchronize();

            // Reorder the seedblock order
            rearrangeSeedQueueWrapper(i, readQueue, writeQueue, d_deleteFlags, d_mergable, numBlocks, gridSize, blockSize);
            hipDeviceSynchronize();

            // Reset mergable flags
            resetMergableFlagsWrapper(d_mergable, numBlocks, gridSize, blockSize);
            hipDeviceSynchronize();
        }

        // Copy results back
        hipMemcpy(hyperBlockMinsC.data(), d_hyperBlockMins, sizeWithoutHBpoints * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(hyperBlockMaxesC.data(), d_hyperBlockMaxes, sizeWithoutHBpoints * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(deleteFlagsC.data(), d_deleteFlags, deleteFlagsC.size() * sizeof(int), hipMemcpyDeviceToHost);
        // Process results
        for (int i = 0; i < hyperBlockMinsC.size(); i += PADDED_LENGTH) {
            
            if (deleteFlagsC[i / PADDED_LENGTH] == -1) continue;  // -1 is a seed block which was merged to. so it doesn't need to be copied back.

            vector<vector<float>> blockMins(FIELD_LENGTH);
            vector<vector<float>> blockMaxes(FIELD_LENGTH);
            for (int j = 0; j < FIELD_LENGTH; j++) {
                blockMins[j].push_back(hyperBlockMinsC[i + j]);
                blockMaxes[j].push_back(hyperBlockMaxesC[i + j]);
            }
            HyperBlock hb(blockMaxes, blockMins, classN);
            resultingBlocks[classN].emplace_back(hb);
        }

        // Free device memory
        hipFree(d_hyperBlockMins);
        hipFree(d_hyperBlockMaxes);
        hipFree(d_combinedMins);
        hipFree(d_combinedMaxes);
        hipFree(d_deleteFlags);
        hipFree(d_points);
        hipFree(d_mergable);
        hipFree(d_seedQueue);
        hipFree(d_writeSeedQueue);
    }

    hyperBlocks.clear();
    for(const vector<HyperBlock>& classBlocks : resultingBlocks) {
      hyperBlocks.insert(hyperBlocks.end(), classBlocks.begin(), classBlocks.end());
    }
}


/**
* This will save the normalized dataset back so that we can use the same one in DV with the same normalization.
*/
void saveNormalizedVersionToCsv(string fileName, vector<vector<vector<float>>>& data) {
    ofstream outFile(fileName);

    if (!outFile.is_open()) {
        cerr << "Error opening file: " << fileName << endl;
        return;
    }

    // Assuming all classes have at least one point, get feature count from the first point of the first class
    int featureCount = data[0][0].size();

    // Write the header
    for (int i = 0; i < featureCount; i++) {
        outFile << "x" << i << ",";
    }
    outFile << "label\n";  // Add label column

    // Iterate through classes
    for (int i = 0; i < data.size(); i++) {
        // Iterate through points in class
        for (int j = 0; j < data[i].size(); j++) {
            // Iterate through attributes of a point
            for (int k = 0; k < data[i][j].size(); k++) {
                outFile << data[i][j][k] << ",";
            }
            outFile << i << "\n";  // Append class label
        }
    }

    outFile.close();
}


/**
* We generate a confusion matrix, but allow for points to fall into multiple blocks at a time
* that is why we go through blocks on outerloop and whole dataset on the inside.
*/
vector<vector<long>> testAccuracyOfHyperBlocks(vector<HyperBlock>& hyperBlocks, vector<vector<vector<float>>> testSet){

  	// Keep track of which points were never inside of a block
    vector<set<int>> pointsNotClassified(CLASS_MAP_TESTING.size(), set<int>(0));
    for(int i = 0; i < CLASS_MAP_TESTING.size(); i++){
    	set.
    }


	// Make a n x n matrix for the confusion matrix
	vector<vector<long>> ultraConfusionMatrix(NUM_CLASSES, vector<long>(NUM_CLASSES, 0));

    cout << "Testing on " << hyperBlocks.size() << " hyperblocks" << endl;
    cout << "Testing on " << testSet.size() << " classes" << endl;
    cout << "Testing on " << testSet[0].size() << " points in first class." << endl;
    cout << "Testing on " << NUM_CLASSES << " classes" << endl;
    cout << "Testing on " << FIELD_LENGTH << " attributes" << endl;

    vector<string> listTraining(NUM_CLASSES);
    for (const auto& entry : CLASS_MAP_TRAINING) {
        int index = entry.second;
        listTraining[index] = entry.first;
    }

    vector<string> listTesting(CLASS_MAP_TESTING.size());
    for (const auto& entry : CLASS_MAP_TESTING) {
        int index = entry.second;
        listTesting[index] = entry.first;
    }


    bool anyPointWasInside = false;

    // Go through all the blocks
	for(int hb = 0; hb < hyperBlocks.size(); hb++){
        HyperBlock& currBlock = hyperBlocks[hb];
        // Go through all the classes in the testSet
		for(int cls = 0; cls < NUM_CLASSES; cls++){
            // go through all the points in a clases
        	for(int pnt = 0; pnt < testSet[cls].size(); pnt++){
           		const vector<float>& point = testSet[cls][pnt];

                if(currBlock.inside_HB(point.size(), point.data())){
                    // Get the actual class name from the training data class index
					string trainingClassName = listTraining[currBlock.classNum];

					// Find what index this class has in the testing data
					int trainingClassIndexInTestingOrder = CLASS_MAP_TESTING[trainingClassName];

					// The testingClassIndex is just cls (the current class we're testing)
					int testingClassIndex = cls;

					ultraConfusionMatrix[testingClassIndex][trainingClassIndexInTestingOrder]++;
                }
                else{
                	// don't know what to put here because it might not have a good
                }
        	}
     	}
    }

	cout << "Any point was inside" << anyPointWasInside <<  endl;
    return ultraConfusionMatrix;
}

/*
void print2DMatrix(vector<vector<long>>& data){
	vector<string> listTraining(NUM_CLASSES);

    for (const auto& entry : CLASS_MAP_TRAINING) {
        int index = entry.second;
        listTraining[index] = entry.first;
    }


  for(const auto& string: listTraining){
    cout << string << "\t" <<endl;
  }

  for(int i = 0; i < data.size(); i++){
	cout << listTraining[i] << "\t"
    for(int j = 0; j < data[i].size(); j++){
      cout << data[i][j] << "\t";
    }
    cout << endl;
  }
}

 */



// Computes the accuracy of the points classifed
void confusionMatrixAccuracy(){


}



void print2DMatrix(vector<vector<long>>& data) {
    vector<string> classLabels(NUM_CLASSES);

    vector<float> accuracies(NUM_CLASSES, 0.0);

    // Calculate the accuracies of each of the rows.
    // Only the diagonal values are correct predictions

for (int i = 0; i < NUM_CLASSES; ++i) {
    long correct = 0;
    long incorrect = 0;
    long totalClassifications = 0;

    for (int j = 0; j < NUM_CLASSES; ++j) {
        totalClassifications += data[i][j];
        if (i == j) {
            correct += data[i][j];  // Diagonal value indicates correct predictions
        } else {
            incorrect += data[i][j];  // Off-diagonal values are incorrect predictions
        }
    }

    if (totalClassifications > 0) {
        accuracies[i] = (float)correct / totalClassifications;
    }
}


    for (const auto& entry : CLASS_MAP_TESTING) {
        int index = entry.second;
        classLabels[index] = entry.first;
    }

    // Calculate column width based on the longest class name and largest number
    size_t maxWidth = 8; // Minimum width

    for (const auto& name : classLabels) {
        maxWidth = max(maxWidth, name.length() + 2);
    }

    for (const auto& row : data) {
        for (const auto& cell : row) {
            string numStr = to_string(cell);
            maxWidth = max(maxWidth, numStr.length() + 2);
        }
    }

    // Print header row with "Actual\Predicted" in the corner
    cout << setw(maxWidth) << "Act\\Pred" << " |";
    for (const auto& name : classLabels) {
        cout << setw(maxWidth) << name << " |";
    }
    cout << endl;

    // Print separator line
    cout << string(maxWidth, '-') << "-+";
    for (size_t i = 0; i < classLabels.size(); i++) {
        cout << string(maxWidth, '-') << "-+";
    }
    cout << endl;

    // Print each row with row label
    for (size_t i = 0; i < data.size(); i++) {
        cout << setw(maxWidth) << classLabels[i] << " |";

        for (size_t j = 0; j < data[i].size(); j++) {
            cout << setw(maxWidth) << data[i][j] << " |";
        }

        cout << accuracies[i] << endl;
    }
}

// Function to clear the console screen (cross-platform)
void clearScreen() {
    #ifdef _WIN32
        system("cls");
    #else
        system("clear");
    #endif
}

// Function to wait for user input before continuing
void waitForEnter() {
    cout << "\nPress Enter to continue...";
    cin.ignore(numeric_limits<streamsize>::max(), '\n');
}

// Function to display the main menu
void displayMainMenu() {
    clearScreen();
    cout << "=== HyperBlock Classification System ===\n\n";
    cout << "1. Import training data.\n";
    cout << "2. Import testing data.\n";
    cout << "3. Save normalized training data.\n";
    cout << endl;
    cout << "4. Import existing hyperblocks.\n";
    cout << "5. Export existing hyperblocks.\n";
    cout << "6. Generate new hyperblocks.\n";
    cout << "7. Simplify hyperblocks.\n";
    cout << "8. Test hyperblocks on dataset.\n";
    cout << endl;
    cout << "9. Exit\n\n";
}


/**
* 	 Assume these for now, implement a better handler later.
*
*    printf("USER WARNING :: ENSURE THAT THERE IS NO ID COLUMN\n");
*    printf("USER WARNING :: ENSURE THAT THE LAST COLUMN IS A CLASS COLUMN\n");
*/
int main(int argc, char* argv[]) {

  	/* File names */
  	string normalizedSaveFile;
	string hyperBlocksImportFileName;
    string trainingDataFileName;
    string testingDataFileName;
	string hyperBlocksExportFileName;

    vector<vector<vector<float>>> testData;
    vector<vector<vector<float>>> trainingData;

    vector<float> minValues(1);	/* Holds the min attribute vals used for normalization */
    vector<float> maxValues(1); /* Holds the max attribute vals used for normalization */

    vector<HyperBlock> hyperBlocks;	/* Store the HyperBlocks we generate or import */


    if(argc == 2){
        COMMAND_LINE_ARGS_CLASS = stoi(argv[1]);
        cout << "Running on class index " << COMMAND_LINE_ARGS_CLASS << endl;
    }

	// In other ML models single point usaully is classified as 1 class only.
    // But in HyperBlocks, a point can be in multiple blocks at the same time
    // thus, we need a more complex matrix to track all of these point classifications.
	vector<vector<long>> ultraConfusionMatrix;

    bool running = true;	// Loop
	int choice;				// Main menu user input "choice"

    // Option to use the command line running style
    if(argc > 1){

    }

    while(running){
       displayMainMenu();
       cin >> choice;
       cin.clear();
       cin.ignore(numeric_limits<streamsize>::max(), '\n');

	   switch (choice) {
          case 1:	// IMPORT TRAINING DATA
			   cout << "Enter training data filename: " << endl;
               system("ls");
               getline(cin, trainingDataFileName);

               // Attempt to read from the file
               trainingData = dataSetup(trainingDataFileName, CLASS_MAP_TRAINING);

                // Reassign them with the correct field length
               minValues.assign(FIELD_LENGTH, std::numeric_limits<float>::infinity());
               maxValues.assign(FIELD_LENGTH, -std::numeric_limits<float>::infinity());
               findMinMaxValuesInDataset(trainingData, minValues, maxValues);

               minMaxNormalization(trainingData, minValues, maxValues);

			   waitForEnter();
               break;

          case 2:	// IMPORT TESTING DATA
		  	 cout << "Enter testing data filename: " << endl;
             system("ls");
             getline(cin, testingDataFileName);

             testData = dataSetup(testingDataFileName, CLASS_MAP_TESTING);
             normalizeTestSet(testData, minValues, maxValues);

             waitForEnter();

		     break;
          case 3:		// SAVE NORMALIZED TRAINING DATA
              cout << "Enter the file to save the normalized training data to: " << endl;

               // Save to the file they input
               getline(cin, normalizedSaveFile);
               saveNormalizedVersionToCsv(normalizedSaveFile, trainingData);

               cout << "Saved normalized training data to: " << normalizedSaveFile << endl;
               waitForEnter();
               break;
          case 4:	// IMPORT EXISTING HYPERBLOCKS
                cout << "Enter existing hyperblocks file name: " << endl;
                getline(cin, hyperBlocksImportFileName);

                hyperBlocks = loadBasicHBsFromCSV(hyperBlocksImportFileName);

                cout << "HyperBlocks imported from file " << hyperBlocksImportFileName << " successfully" << endl;
                waitForEnter();
                break;
          case 5: // EXPORT HYPERBLOCKS
            	cout << "Enter the file to save HyperBlocks to: " << endl;

               // Save to the file they input
               getline(cin, hyperBlocksExportFileName);


			   saveBasicHBsToCSV(hyperBlocks, hyperBlocksExportFileName);
               break;
          case 6:	// GENERATE NEW HYPERBLOCKS
                if (trainingData.empty()) {
                    cout << "\nError: Please import training data first." << endl;
                    waitForEnter();
                } else {
                    hyperBlocks.clear();
                    generateHBs(trainingData, hyperBlocks);
                }
                cout << "Finished Generating HyperBlocks" << endl;

                waitForEnter();
                break;
          case 7:		// SIMPLIFY HYPERBLOCKS
               cout << "Simplifications Interface Not Yet Implemented. " << endl;
               //hyperBlocks = simplifyExistingHyperBlocks(hyperBlocks);
               break;
          case 8:		// TEST HYPERBLOCKS ON DATASET
             cout << "Testing hyperblocks on testing dataset" << endl;
             ultraConfusionMatrix = testAccuracyOfHyperBlocks(hyperBlocks, testData);
             print2DMatrix(ultraConfusionMatrix);
             waitForEnter();
             break;
          case 9:		// EXIT
             running = false;
             break;
          default:
             cout << "\nInvalid choice. Please try again." << endl;
             waitForEnter();
             break;
        }
	}
    return 0;
}
